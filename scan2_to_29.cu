#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// This code is going to support multiple blocked array scan
// I mean an array of size 1 to 1 << 29 elements

/*
    for sizes 1<<28 and 1<<29, you cannot copy the whole array from cpu to gpu!
    it does not fit in our gpu global memory!
    with "checkDeviceInfor.cu" I checked gpu global memory :
    Total amount of global memory: 1007747072 bytes
    which is around 1 Gbytes!
    now for an array of size 1<<27 floats we have
    1<<27 * 4 bytes / (1<<20) = 512 Mbytes!
    so for array size 1<<28, it reaches 1.024 Gbytes and 
    therefore it does not fit into the gpu mem.

    So I will make the size 1<<27 my standard size, and break bigger
    arrays into this size!
    for eample for an array with size 1<<29, I should bring 4 segments
    of size 1<<27 of the array to gpy, scan them and do some little tricks
    to be able to do that.

*/


// I performed in-place scan, meaning I scanned the array in itself, not a new array
// for the sake of mem

#include "scan2.h"
#include "gpuerrors.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

// for arrays of size : 1 to 1024 elements
__global__ void scan_kernel(float* ad, float* cd, int n)
{
    __shared__ float sd[1024]; // shared mem -> has to be fixed sized, if it was possible
    // it was better to write sd[n] but since we are not able, I allocated the max size possible in
    // this code which is 1024 
    
    //int i = bx * n + tx; // global indexing
    int j = tx; // local indexing
    int k = bx * n / 2 + tx;

    // since the input array is of size n and we have n / 2 threads, each thread 
    // will bring 2 elements.
    // actually in this code since we have only 1 block, bx = 0
    // and j and k are equal.
    sd[2 * j] = ad[2 * k];
    sd[2 * j + 1] = ad[2 * k + 1];
    //__syncthreads(); // wait until the sh mem of each block is full
    // actually we do not need this syncthread, since each thread is bringing 
    // the elements it needs in the first step of the loop
    // ex : thread 4 brings ad[8], ad[9]
    // and thread 4 needs only ad[8], ad[9] in the first step(first iteration of for) 
    int d;
    for(d = 1; d <= n / 2; d = d * 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            
        }
        __syncthreads(); // each thread has to wait here till all other threads reach here
        // then go to the next step(next iteration).
    }
    // now that the first half is done : 

    
    // this part is for inclusive scan!
    //uncomment if you want inclusive scan
    /////////////// very important!!!!!!!!!!!!!! spare variabele! each thread has it's own!
    //float spare; // for inclusive scan
    //if(j == (n / 2) - 1) // j = n/2 - 1 is the last thread, because we have n / 2 threads
    //{
    //    //float spare; // do not know why it raises an error if declared here
    //    spare = sd[n - 1]; // here spare of the last thread gets the valus only! do not expect other threads
    //    // to see any change in their spare variable!!!    
    //}
    //__syncthreads(); // VERY important! o.w it may result in wrong answer!
    //float spare; // keep it for inclusive scan
    //spare = sd[n - 1];
    


    if(j == 0) // thread number 0 or the first thread sets the last element to zero.
    {
        //spare = sd[n - 1];
        sd[n - 1] = 0;
    }
    
    // very very important note here; 
    // if you do j == anything other than 0
    // you will need __syncthreads() !
    //__syncthreads(); so we do not need sync here!!
    
    float temp; // each thread has it's own temp
    for(d = n / 2; d >= 1; d = d / 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            temp = sd[(2 * d) * (j + 1) - 1];
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            sd[d * (2 * j + 1) - 1] = temp;
        }
        __syncthreads();
    }
    // each thread copies 2 elements from sh mem to global mem
    
    
        //uncomment this part for exclusive scan
        //exclusive scan
    cd[2 * j] = sd[2 * j];
    cd[2 * j + 1] = sd[2 * j + 1];
    

    
    //inclusive scan
    // since fisrt element of the answer is 0 in belleloch algo
    //we should not keep the first element of the answer in inclusive scan.  
    //if(j == (n / 2) - 1) // again the last thread! should load back the it's spare variable 
    //// into the last element of the answer array
    //{
    //    cd[n - 2] = sd[n - 1]; 
    //    cd[n - 1] = spare; 
    //}
    //else
    //{
    //    cd[2 * j] = sd[2 * j + 1];
    //    cd[2 * j + 1] = sd[2 * j + 2];
    //}   
}

__global__ void scan_kernel_inclusive(float* ad, float* cd, int n)
{
    __shared__ float sd[1024]; // shared mem -> has to be fixed sized, if it was possible
    // it was better to write sd[n] but since we are not able, I allocated the max size possible in
    // this code which is 1024 
    
    //int i = bx * n + tx; // global indexing
    int j = tx; // local indexing
    int k = bx * n / 2 + tx;

    // since the input array is of size n and we have n / 2 threads, each thread 
    // will bring 2 elements.
    // actually in this code since we have only 1 block, bx = 0
    // and j and k are equal.
    sd[2 * j] = ad[2 * k];
    sd[2 * j + 1] = ad[2 * k + 1];
    //__syncthreads(); // wait until the sh mem of each block is full
    // actually we do not need this syncthread, since each thread is bringing 
    // the elements it needs in the first step of the loop
    // ex : thread 4 brings ad[8], ad[9]
    // and thread 4 needs only ad[8], ad[9] in the first step(first iteration of for) 
    int d;
    for(d = 1; d <= n / 2; d = d * 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            
        }
        __syncthreads(); // each thread has to wait here till all other threads reach here
        // then go to the next step(next iteration).
    }
    // now that the first half is done : 

    
    // this part is for inclusive scan!
    //uncomment if you want inclusive scan
    /////////////// very important!!!!!!!!!!!!!! spare variabele! each thread has it's own!
    float spare; // for inclusive scan
    if(j == (n / 2) - 1) // j = n/2 - 1 is the last thread, because we have n / 2 threads
    {
        //float spare; // do not know why it raises an error if declared here
        spare = sd[n - 1]; // here spare of the last thread gets the valus only! do not expect other threads
        // to see any change in their spare variable!!!    
    }
    __syncthreads(); // VERY important! o.w it may result in wrong answer!
    //float spare; // keep it for inclusive scan
    //spare = sd[n - 1];
    


    if(j == 0) // thread number 0 or the first thread sets the last element to zero.
    {
        //spare = sd[n - 1];
        sd[n - 1] = 0;
    }
    
    // very very important note here; 
    // if you do j == anything other than 0
    // you will need __syncthreads() !
    //__syncthreads(); so we do not need sync here!!
    
    float temp; // each thread has it's own temp
    for(d = n / 2; d >= 1; d = d / 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            temp = sd[(2 * d) * (j + 1) - 1];
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            sd[d * (2 * j + 1) - 1] = temp;
        }
        __syncthreads();
    }
    // each thread copies 2 elements from sh mem to global mem
    
    
        //uncomment this part for exclusive scan
        //exclusive scan
    //cd[2 * j] = sd[2 * j];
    //cd[2 * j + 1] = sd[2 * j + 1];
    

    
    //inclusive scan
    // since fisrt element of the answer is 0 in belleloch algo
    //we should not keep the first element of the answer in inclusive scan.  
    if(j == (n / 2) - 1) // again the last thread! should load back the it's spare variable 
    // into the last element of the answer array
    {
        cd[n - 2] = sd[n - 1]; 
        cd[n - 1] = spare; 
    }
    else
    {
        cd[2 * j] = sd[2 * j + 1];
        cd[2 * j + 1] = sd[2 * j + 2];
    }   
}

__global__ void scan_each_block_kernel_inc(float* ad, float* cd, float* weight, int n)
{
    // in this kernel, 1024s can be replaced by n
    // 512s -> by n/2
    // 1023s -> by n - 1
    // 511s -> by n / 2 - 1
    // I just did not do this for the readability of code.
    __shared__ float sd[1024]; // shared mem
    int i = bx * 1024 + tx; // global indexing
    int j = tx; // local indexing
    int k = bx * 512 + tx; // global indexing


    sd[2 * j] = ad[2 * k];
    sd[2 * j + 1] = ad[2 * k + 1];
    //__syncthreads(); // wait until the sh mem of each block is full
    // actually we do not need this syncthread, since each thread is bringing 
    // the elements it needs in the first step of the loop
    // ex : thread 4 of block 0 brings ad[8], ad[9]
    // and thread 4 needs only sd[8], sd[9] in the first step(first iteration of for)
    int d;
    for(d = 1; d <= n / 2; d = d * 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            
        }
        __syncthreads();
    }
    // now that the first half is done : 

    
    // this part is for inclusive scan!
    //uncomment if you want inclusive scan
    /////////////// very important!!!!!!!!!!!!!! spare variabele! each thread has it's own!
    float spare; // for inclusive scan
    if(j == 511) // last thread of each block
    {
        //float spare; // do not know why it raises an error if declared here
        spare = sd[1023]; // here spare of thread 511 gets the valus only! do not expect other threads
        // to see any change in their spare variable!!!
        weight[bx] = sd[1023]; // for the next kernel func when we want to compensate scans of blocks             
    }
    __syncthreads(); // VERY important! o.w it may result in wrong answer!
    //float spare; // keep it for inclusive scan
    //spare = sd[1023];


    if(j == 0) // thread number 0 of each block sets the last element to zero.
    {
        //spare = sd[1023];
        sd[1023] = 0;
    }
    
    // very very important note here; 
    // if you do j == anything other than 0
    // you will need __syncthreads() !
    //__syncthreads(); so we do not need sync here!!
    
    float temp; // each thread has it's own temp
    for(d = n / 2; d >= 1; d = d / 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            temp = sd[(2 * d) * (j + 1) - 1];
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            sd[d * (2 * j + 1) - 1] = temp;
        }
        __syncthreads();
    }
    // each thread copies 2 elements from sh mem to global mem
    
    /*
        //uncomment this part for exclusive scan
        //exclusive scan
    cd[2 * k] = sd[2 * j];
    cd[2 * k + 1] = sd[2 * j + 1];
    */

    
    //inclusive scan
    // since first element of the answer is 0 in belleloch algo
    //we should not keep the first element of the answer in inclusive scan.  
    if(j == 511)
    {
        cd[bx * 1024 + 1022] = sd[1023];
        cd[bx * 1024 + 1023] = spare; 
    }
    else
    {
        cd[2 * k] = sd[2 * j + 1];
        cd[2 * k + 1] = sd[2 * j + 2];
    }  

}

__global__ void scan_each_block_kernel_inc_26_27(float* ad, float* cd, float* weight, int n)
{
    // in this kernel, 1024s can be replaced by n
    // 512s -> by n/2
    // 1023s -> by n - 1
    // 511s -> by n / 2 - 1
    // I just did not do this for the readability of code.
    __shared__ float sd[1024]; // shared mem
    int i = bx * 1024 + tx; // global indexing
    int j = tx; // local indexing
    //int k = bx * 512 + tx; // global indexing
    int k = by * ((1<<25)>>1) + bx * 512 + tx;

    // each block has 1024 elements and 512 threads, so each thread brings 2 elements
    sd[2 * j] = ad[2 * k];
    sd[2 * j + 1] = ad[2 * k + 1];
    //__syncthreads(); // wait until the sh mem of each block is full
    // actually we do not need this syncthread, since each thread is bringing 
    // the elements it needs in the first step of the loop
    // ex : thread 4 of block 0 brings ad[8], ad[9]
    // and thread 4 needs only sd[8], sd[9] in the first step(first iteration of for)
    int d;
    for(d = 1; d <= n / 2; d = d * 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            
        }
        __syncthreads();
    }
    // now that the first half is done : 

    
    // this part is for inclusive scan!
    //uncomment if you want inclusive scan
    /////////////// very important!!!!!!!!!!!!!! spare variabele! each thread has it's own!
    float spare; // for inclusive scan
    if(j == 511) // last thread of each block
    {
        //float spare; // do not know why it raises an error if declared here
        spare = sd[1023]; // here spare of thread 511 gets the valus only! do not expect other threads
        // to see any change in their spare variable!!!
        weight[by * (1<<15) + bx] = sd[1023]; // for the next kernel func when we want to compensate scans of blocks             
    }
    __syncthreads(); // VERY important! o.w it may result in wrong answer!
    //float spare; // keep it for inclusive scan
    //spare = sd[1023];


    if(j == 0) // thread number 0 of each block sets the last element to zero.
    {
        //spare = sd[1023];
        sd[1023] = 0;
    }
    
    // very very important note here; 
    // if you do j == anything other than 0
    // you will need __syncthreads() !
    //__syncthreads(); so we do not need sync here!!
    
    float temp; // each thread has it's own temp
    for(d = n / 2; d >= 1; d = d / 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            temp = sd[(2 * d) * (j + 1) - 1];
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            sd[d * (2 * j + 1) - 1] = temp;
        }
        __syncthreads();
    }
    // each thread copies 2 elements from sh mem to global mem
    
    /*
        //uncomment this part for exclusive scan
        //exclusive scan
    cd[2 * k] = sd[2 * j];
    cd[2 * k + 1] = sd[2 * j + 1];
    */

    
    //inclusive scan
    // since first element of the answer is 0 in belleloch algo
    //we should not keep the first element of the answer in inclusive scan.  
    if(j == 511)
    {
        //////////////////////////////////////////////////VERYYYYYYYYYYYYYYYYYYYYYYYYYYYY
        // BY * 1<<25
        cd[by * (1<<25) + bx * 1024 + 1022] = sd[1023];
        cd[by * (1<<25) + bx * 1024 + 1023] = spare; 
    }
    else
    {
        cd[2 * k] = sd[2 * j + 1];
        cd[2 * k + 1] = sd[2 * j + 2];
    }  

}

__global__ void scan_each_block_kernel_exc(float* ad, float* cd, float* weight, int n)
{
    // in this kernel, 1024s can be replaced by n
    // 512s -> by n/2
    // 1023s -> by n - 1
    // 511s -> by n / 2 - 1
    // I just did not do this for the readability of code.
    __shared__ float sd[1024]; // shared mem
    int i = bx * 1024 + tx; // global indexing
    int j = tx; // local indexing
    int k = bx * 512 + tx; // global indexing


    sd[2 * j] = ad[2 * k];
    sd[2 * j + 1] = ad[2 * k + 1];
    //__syncthreads(); // wait until the sh mem of each block is full
    // actually we do not need this syncthread, since each thread is bringing 
    // the elements it needs in the first step of the loop
    // ex : thread 4 of block 0 brings ad[8], ad[9]
    // and thread 4 needs only sd[8], sd[9] in the first step(first iteration of for)
    int d;
    for(d = 1; d <= n / 2; d = d * 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            
        }
        __syncthreads();
    }
    // now that the first half is done : 

    
    // this part is for inclusive scan!
    //uncomment if you want inclusive scan
    /////////////// very important!!!!!!!!!!!!!! spare variabele! each thread has it's own!
   // float spare; // for inclusive scan
    if(j == 511) // last thread of each block
    {
   //     //float spare; // do not know why it raises an error if declared here
   //     spare = sd[1023]; // here spare of thread 511 gets the valus only! do not expect other threads
   //     // to see any change in their spare variable!!!
        weight[bx] = sd[1023]; // for the next kernel func when we want to compensate scans of blocks             
    }
   __syncthreads(); // VERY important! o.w it may result in wrong answer!
    //float spare; // keep it for inclusive scan
    //spare = sd[1023];


    if(j == 0) // thread number 0 of each block sets the last element to zero.
    {
        //spare = sd[1023];
        sd[1023] = 0;
    }
    
    // very very important note here; 
    // if you do j == anything other than 0
    // you will need __syncthreads() !
    //__syncthreads(); so we do not need sync here!!
    
    float temp; // each thread has it's own temp
    for(d = n / 2; d >= 1; d = d / 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            temp = sd[(2 * d) * (j + 1) - 1];
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            sd[d * (2 * j + 1) - 1] = temp;
        }
        __syncthreads();
    }
    // each thread copies 2 elements from sh mem to global mem
    
    
        //uncomment this part for exclusive scan
        //exclusive scan
    cd[2 * k] = sd[2 * j];
    cd[2 * k + 1] = sd[2 * j + 1];
    

    
    //inclusive scan
    // since first element of the answer is 0 in belleloch algo
    //we should not keep the first element of the answer in inclusive scan.  
  //  if(j == 511)
  //  {
  //      cd[bx * 1024 + 1022] = sd[1023];
  //      cd[bx * 1024 + 1023] = spare; 
  //  }
  //  else
  //  {
  //      cd[2 * k] = sd[2 * j + 1];
  //      cd[2 * k + 1] = sd[2 * j + 2];
  //  }  

}


__global__ void scan_weight_kernel(float* ad, float* cd, int n)
{
    __shared__ float sd[1024]; // shared mem
    int i = bx * n + tx; // global indexing
    int j = tx; // local indexing
    int k = bx * n / 2 + tx; // global indexing

    sd[2 * j] = ad[2 * k];
    sd[2 * j + 1] = ad[2 * k + 1];
    //__syncthreads(); // wait until the sh mem of each block is full
    
    int d;
    for(d = 1; d <= n / 2; d = d * 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            
        }
        __syncthreads();
    }
    // now that the first half is done : 

    /*
    // this part is for inclusive scan!
    //uncomment if you want inclusive scan
    /////////////// very important!!!!!!!!!!!!!! spare variabele! each thread has it's own!
    float spare; // for inclusive scan
    if(j == n / 2 - 1) // last thread of each block
    {
        //float spare; // do not know why it raises an error if declared here
        spare = sd[n - 1]; // here spare of thread 511 gets the valus only! do not expect other threads
        // to see any change in their spare variable!!!             
    }
    __syncthreads(); // VERY important! o.w it may result in wrong answer!
    //float spare; // keep it for inclusive scan
    //spare = sd[n - 1];
    */

    if(j == 0) // thread number 0 of each block sets the last element to zero.
    {
        //spare = sd[1023];
        sd[n - 1] = 0;
    }
    
    // very very important note here; 
    // if you do j == anything other than 0
    // you will need __syncthreads() !
    //__syncthreads(); so we do not need sync here!!
    
    float temp; // each thread has it's own temp
    for(d = n / 2; d >= 1; d = d / 2)
    {
        if(j < n / (2 * d))
        {
            //sd[(2 * d) * j + d - 1 + d] = sd[(2 * d) * j + d - 1] + sd[(2 * d) * j + d - 1 + d]; this is equal to the next line
            temp = sd[(2 * d) * (j + 1) - 1];
            sd[(2 * d) * (j + 1) - 1] = sd[d * (2 * j + 1) - 1] + sd[(2 * d) * (j + 1) - 1];
            sd[d * (2 * j + 1) - 1] = temp;
        }
        __syncthreads();
    }
    // each thread copies 2 elements from sh mem to global mem
    
    
        //uncomment this part for exclusive scan
        //exclusive scan
    cd[2 * k] = sd[2 * j];
    cd[2 * k + 1] = sd[2 * j + 1];
    

    /*
    //inclusive scan
    // since first element of the answer is 0 in belleloch algo
    //we should not keep the first element of the answer in inclusive scan.  
    if(j == n / 2 - 1)
    {
        cd[bx * n + n - 2] = sd[n - 1];
        cd[bx * n + n - 1] = spare; 
    }
    else
    {
        cd[2 * k] = sd[2 * j + 1];
        cd[2 * k + 1] = sd[2 * j + 2];
    }  
    */
}

__global__ void add_weight_kernel(float* cd, float* sweight)
{
   // __shared__ float sd[1024]; // shared mem
   // __shared__ float sw[1024];
    int i = bx * 1024 + tx; // global indexing
    int j = tx; // local indexing
    int k = bx * 512 + tx; // global indexing

   // sd[j] = cd[i];
   // sw[j] = sweight[j];
   // __syncthreads(); // wait until the sh mem of each block is full

   // sd[j] = sd[j] + sw[bx];
   // cd[i] = sd[j];
    cd[i] = cd[i] + sweight[bx];
}


__global__ void add_weight_kernel_26_27(float* cd, float* sweight, float lastpartans)
{
   // __shared__ float sd[1024]; // shared mem
   // __shared__ float sw[1024];
    int i = by * (1<<25) + bx * 1024 + tx; // global indexing
    int j = tx; // local indexing
    int k = bx * 512 + tx; // global indexing

   // sd[j] = cd[i];
   // sw[j] = sweight[j];
   // __syncthreads(); // wait until the sh mem of each block is full

   // sd[j] = sd[j] + sw[bx];
   // cd[i] = sd[j];
    cd[i] = cd[i] + sweight[by * (1<<15) + bx] + lastpartans;
}
////////////////////////////////////////////////////////
////////////////////////////////////////////////////////
// n is the size of the array which is n = 1 << M
void gpuKernel(float* a, float* c,int n) 
{
    const int numberOfArrayElementsPerBlock = 1024;
    float* ad;
	//float* cd;
    float* weight;
    float* newweight;
    float lastpartans = 0; // this is allocated in cpu mem
    //float* sweight;
    // allocate gpu side pointers

    // if the size of the array is in range(1,2,4,...to 1024)-------------------small array---------------
    if(n <= 1024)
    {
        HANDLE_ERROR(hipMalloc((void**)&ad, n * sizeof(float)));
        HANDLE_ERROR(hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice));
        int threads = n / 2;
        scan_kernel_inclusive <<<1, threads>>> (ad, ad, n);
        HANDLE_ERROR(hipMemcpy(c, ad, n * sizeof(float), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipFree(ad));
        return;
    }
    //...............................................................................................

    if(n >= (1<<28))
    {
        int iter = n / (1<<27); // number of times to perform 1<<27 sized scan serially
        int standardsize = (1<<27);
        HANDLE_ERROR(hipMalloc((void**)&ad, standardsize * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void**)&weight, (standardsize / numberOfArrayElementsPerBlock) * sizeof(float))); // let us not over allocate it!
        HANDLE_ERROR(hipMalloc((void**)&newweight, (standardsize / (numberOfArrayElementsPerBlock * numberOfArrayElementsPerBlock)) * sizeof(float)));

        for(int i = 0; i < iter; i++)
        {
            HANDLE_ERROR(hipMemcpy(ad, a + i * standardsize, standardsize * sizeof(float), hipMemcpyHostToDevice));
            int blocks = standardsize / numberOfArrayElementsPerBlock; // number of blocks
            int threads;
            threads = numberOfArrayElementsPerBlock / 2; // 512
            int division = standardsize / (1<<25);
            dim3 dimGrid(blocks / division, division);
            scan_each_block_kernel_inc_26_27 <<<dimGrid, threads>>> (ad, ad, weight, numberOfArrayElementsPerBlock); 
            ///////////----SCAN WEIGHT----////////////////////////////////////////////////////////
            int weightArraySize = blocks;
            if(weightArraySize > 1024)
            {
                blocks = weightArraySize / numberOfArrayElementsPerBlock; // because the new weight array size 
                // has changed to weightArraySize, so recalc blocks
                scan_each_block_kernel_exc <<<blocks, threads>>> (weight, weight, newweight, numberOfArrayElementsPerBlock);
                int newweightArraySize = blocks;
                threads = newweightArraySize / 2;
                scan_weight_kernel <<<1, threads>>>(newweight, newweight, newweightArraySize);
                // now add the bias 
                add_weight_kernel <<<blocks, numberOfArrayElementsPerBlock>>>(weight, newweight);
            }
            else
            {
                threads = weightArraySize / 2; // like reduce, for an array of size n I will launch n / 2 threads.
                //int blocks = n / maxThreadsPerBlock; // number of blocks
                // BLOCKS = 1 SINCE I HAVE WRITTEN THIS CODE ONLY FOR SIZE 2 to 1024
                scan_kernel <<<1, threads>>> (weight, weight, weightArraySize);
            }
            //////////////////////////////////////////////////////////////////
            blocks = standardsize / numberOfArrayElementsPerBlock; // remember the number of blocks on ad arrays
            add_weight_kernel_26_27 <<<dimGrid, numberOfArrayElementsPerBlock>>>(ad, weight, lastpartans);
            HANDLE_ERROR(hipMemcpy(c + i * standardsize, ad, standardsize * sizeof(float), hipMemcpyDeviceToHost));
            lastpartans = c[(i + 1) * standardsize - 1];    
        }

        HANDLE_ERROR(hipFree(ad));
        //HANDLE_ERROR(hipFree(cd));
        //HANDLE_ERROR(hipFree(sweight));
        HANDLE_ERROR(hipFree(weight));
        HANDLE_ERROR(hipFree(newweight));
        return;
    }


    HANDLE_ERROR(hipMalloc((void**)&ad, n * sizeof(float)));
    //HANDLE_ERROR(hipMalloc((void**)&cd, n * sizeof(float)));

    //HANDLE_ERROR(hipMalloc((void**)&weight, n * sizeof(float))); // overallocated it
    HANDLE_ERROR(hipMalloc((void**)&weight, (n / numberOfArrayElementsPerBlock) * sizeof(float))); // let us not over allocate it!
    HANDLE_ERROR(hipMalloc((void**)&newweight, (n / (numberOfArrayElementsPerBlock * numberOfArrayElementsPerBlock)) * sizeof(float)));
    // weight will have n / 1024 elements, because each block sends it's reduced ans
    // to weight and we have n / 1024 blocks.
    // copy input from cpu mem to gpu mem
    HANDLE_ERROR(hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice));

    int blocks = n / numberOfArrayElementsPerBlock; // number of blocks
    const int maxThreadsPerBlock = 1024;
    int threads;
    if(n >= (1<<26))
    {
        threads = numberOfArrayElementsPerBlock / 2; // 512
        int division = n / (1<<25);
        dim3 dimGrid(blocks / division, division);
        scan_each_block_kernel_inc_26_27 <<<dimGrid, threads>>> (ad, ad, weight, numberOfArrayElementsPerBlock);    
    }
    else
    {
        threads = numberOfArrayElementsPerBlock / 2; // 512
        scan_each_block_kernel_inc <<<blocks, threads>>> (ad, ad, weight, numberOfArrayElementsPerBlock); // ad -> scan -> cd
        // now let us scan the weight array to add the appropriate number to each scanned input block
        // size of the weight array is equal to the number of intial blocks,
        // because each block, has sent it's reduced answer to one element in weight array 
    }
    //////////////////////////////////WEIGHT SCAN////////////////////////////////////////////////////
    int weightArraySize = blocks;
    if(weightArraySize > 1024)
    {
        blocks = weightArraySize / numberOfArrayElementsPerBlock; // because the new weight array size 
        // has changed to weightArraySize, so recalc blocks
        scan_each_block_kernel_exc <<<blocks, threads>>> (weight, weight, newweight, numberOfArrayElementsPerBlock);
        int newweightArraySize = blocks;
        threads = newweightArraySize / 2;
        scan_weight_kernel <<<1, threads>>>(newweight, newweight, newweightArraySize); // weight -> scan -> sweight
        // now add the bias 
        add_weight_kernel <<<blocks, numberOfArrayElementsPerBlock>>>(weight, newweight);
    }
    else
    {
        threads = weightArraySize / 2; // like reduce, for an array of size n I will launch n / 2 threads.
        //int blocks = n / maxThreadsPerBlock; // number of blocks
        // BLOCKS = 1 SINCE I HAVE WRITTEN THIS CODE ONLY FOR SIZE 2 to 1024
        scan_kernel <<<1, threads>>> (weight, weight, weightArraySize);
    }
    /////////////////////////////////////////////////////////////////////////////////////////////////
    //////////////////ADD your block scanned ad to the scanned weight elements///////////
    if(n >= (1<<26))
    {
        blocks = n / numberOfArrayElementsPerBlock; // remember the number of blocks on ad arrays
        int division = n / (1<<25);
        dim3 dimGrid(blocks / division, division);
        add_weight_kernel_26_27 <<<dimGrid, numberOfArrayElementsPerBlock>>>(ad, weight,0);
    }
    else
    {
        blocks = n / numberOfArrayElementsPerBlock; // remember the number of blocks on ad arrays
        add_weight_kernel <<<blocks, numberOfArrayElementsPerBlock>>>(ad, weight);
    }
    ////////////////////////////////////////////////////////////////////////////////////
    HANDLE_ERROR(hipMemcpy(c, ad, n * sizeof(float), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(ad));
    //HANDLE_ERROR(hipFree(cd));
    //HANDLE_ERROR(hipFree(sweight));
    HANDLE_ERROR(hipFree(weight));
    HANDLE_ERROR(hipFree(newweight));
}